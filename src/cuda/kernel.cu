#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addKernel(int *d_a, int *d_b, int *d_c, int N) {
    int i = threadIdx.x;
    if (i < N) {
        d_c[i] = d_a[i] + d_b[i];
    }
}

extern "C" void launch_add_kernel(int *a, int *b, int *c, int N) {
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N * sizeof(int));
    hipMalloc((void **)&d_b, N * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, N>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}